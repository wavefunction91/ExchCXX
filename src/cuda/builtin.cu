#include "hip/hip_runtime.h"
/**
 * ExchCXX Copyright (c) 2020-2022, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * (1) Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 * 
 * (2) Redistributions in binary form must reproduce the above copyright
 * notice, this list of conditions and the following disclaimer in the
 * documentation and/or other materials provided with the distribution.
 * 
 * (3) Neither the name of the University of California, Lawrence Berkeley
 * National Laboratory, U.S. Dept. of Energy nor the names of its contributors
 * may be used to endorse or promote products derived from this software
 * without specific prior written permission.
 * 
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * 
 * You are under no obligation whatsoever to provide any bug fixes, patches,
 * or upgrades to the features, functionality or performance of the source
 * code ("Enhancements") to anyone; however, if you choose to make your
 * Enhancements available either publicly, or directly to Lawrence Berkeley
 * National Laboratory, without imposing a separate written license agreement
 * for such Enhancements, then you hereby grant the following license: a
 * non-exclusive, royalty-free perpetual license to install, use, modify,
 * prepare derivative works, incorporate into other computer software,
 * distribute, and sublicense such enhancements or derivative works thereof,
 * in binary and source code form.
 */

#include <exchcxx/impl/builtin/kernel_type.hpp>
#include <exchcxx/impl/builtin/util.hpp>
#include <exchcxx/impl/builtin/kernels.hpp>
#include <exchcxx/util/div_ceil.hpp>

namespace ExchCXX {
namespace detail {


template <typename KernelType>
__global__ LDA_EXC_GENERATOR( device_eval_exc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    traits::eval_exc_unpolar( rho[tid], eps[tid] );

  }

}

template <typename KernelType>
__global__ LDA_EXC_GENERATOR( device_eval_exc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto rho_i = rho + 2*tid;
    traits::eval_exc_polar( rho_i[0], rho_i[1], eps[tid] );

  }

}

template <typename KernelType>
__global__ LDA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    traits::eval_exc_vxc_unpolar( rho[tid], eps[tid], vxc[tid] );

  }

}

template <typename KernelType>
__global__ LDA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto rho_i = rho + 2*tid;
    auto vxc_i = vxc + 2*tid;

    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], eps[tid], 
      vxc_i[0], vxc_i[1] );

  }

}

template <typename KernelType>
__global__ LDA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e;
  if( tid < N ) {

    traits::eval_exc_unpolar( rho[tid], e );
    eps[tid] += scal_fact * e;

  }

}

template <typename KernelType>
__global__ LDA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto rho_i = rho + 2*tid;

    double e;
    traits::eval_exc_polar( rho_i[0], rho_i[1], e );
    
    eps[tid] += scal_fact * e;

  }

}

template <typename KernelType>
__global__ LDA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e,v;
  if( tid < N ) {

    traits::eval_exc_vxc_unpolar( rho[tid], e, v );
    eps[tid] += scal_fact * e;
    vxc[tid] += scal_fact * v;

  }

}

template <typename KernelType>
__global__ LDA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto rho_i = rho + 2*tid;
    auto vxc_i = vxc + 2*tid;

    double v_a, v_b, e;
    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], e, v_a, v_b);
    eps[tid] += scal_fact * e;
    vxc_i[0] += scal_fact * v_a;
    vxc_i[1] += scal_fact * v_b;

  }

}

template <typename KernelType>
__global__ GGA_EXC_GENERATOR( device_eval_exc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    traits::eval_exc_unpolar( rho[tid], sigma[tid], eps[tid] );

  }

}

template <typename KernelType>
__global__ GGA_EXC_GENERATOR( device_eval_exc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;

    traits::eval_exc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], eps[tid] );

  }

}

template <typename KernelType>
__global__ GGA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    traits::eval_exc_vxc_unpolar( rho[tid], sigma[tid], eps[tid], 
      vrho[tid], vsigma[tid] );

  }

}

template <typename KernelType>
__global__ GGA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i    = rho   + 2*tid;
    auto* sigma_i  = sigma + 3*tid;
    auto* vrho_i   = vrho   + 2*tid;
    auto* vsigma_i = vsigma + 3*tid;

    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], eps[tid], vrho_i[0], vrho_i[1],
      vsigma_i[0], vsigma_i[1], vsigma_i[2] );

  }

}


template <typename KernelType>
__global__ GGA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e;
  if( tid < N ) {
                                      
    traits::eval_exc_unpolar( rho[tid], sigma[tid], e );
    eps[tid] += scal_fact * e;
     
  }

}

template <typename KernelType>
__global__ GGA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;
    double e;
    traits::eval_exc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], e );
    eps[tid] += scal_fact * e;
     

  }

}

template <typename KernelType>
__global__ GGA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e, vr, vs;
  if( tid < N ) {

    traits::eval_exc_vxc_unpolar( rho[tid], sigma[tid], e, vr, vs );
    eps[tid]    += scal_fact * e;
    vrho[tid]   += scal_fact * vr;
    vsigma[tid] += scal_fact * vs;

  }

}

template <typename KernelType>
__global__ GGA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i    = rho   + 2*tid;
    auto* sigma_i  = sigma + 3*tid;
    auto* vrho_i   = vrho   + 2*tid;
    auto* vsigma_i = vsigma + 3*tid;
                                                         
    double e, vra, vrb, vsaa,vsab,vsbb;
    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], e, vra, vrb, vsaa, vsab, vsbb );

    eps[tid]    += scal_fact * e;
    vrho_i[0]   += scal_fact * vra;
    vrho_i[1]   += scal_fact * vrb;
    vsigma_i[0] += scal_fact * vsaa;
    vsigma_i[1] += scal_fact * vsab;
    vsigma_i[2] += scal_fact * vsbb;

  }

}


template <typename KernelType>
__global__ MGGA_EXC_GENERATOR( device_eval_exc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    const double lapl_use  = traits::needs_laplacian ? lapl[tid] : 0.0;
    traits::eval_exc_unpolar( rho[tid], sigma[tid], lapl_use, tau[tid], eps[tid] );

  }

}


template <typename KernelType>
__global__ MGGA_EXC_GENERATOR( device_eval_exc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;
    auto* lapl_i  = traits::needs_laplacian ? (lapl + 2*tid) : nullptr;
    auto* tau_i   = tau   + 2*tid;

    const double lapl_a_use = traits::needs_laplacian ? lapl_i[0] : 0.0;
    const double lapl_b_use = traits::needs_laplacian ? lapl_i[1] : 0.0;

    traits::eval_exc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], lapl_a_use, lapl_b_use, tau_i[0],
      tau_i[1], eps[tid] );

  }

}

template <typename KernelType>
__global__ MGGA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    const double lapl_use  = traits::needs_laplacian ? lapl[tid] : 0.0;

    double dummy;
    auto& vlapl_return = traits::needs_laplacian ? vlapl[tid] : dummy; 
    traits::eval_exc_vxc_unpolar( rho[tid], sigma[tid], lapl_use, tau[tid],
      eps[tid], vrho[tid], vsigma[tid], vlapl_return, vtau[tid] );

  }

}

template <typename KernelType>
__global__ MGGA_EXC_VXC_GENERATOR( device_eval_exc_vxc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double dummy_vlapl[2];

  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;
    auto* lapl_i  = traits::needs_laplacian ? (lapl + 2*tid) : lapl;
    auto* tau_i   = tau   + 2*tid;

    auto* vrho_i   = vrho   + 2*tid;
    auto* vsigma_i = vsigma + 3*tid;
    auto* vlapl_i  = traits::needs_laplacian ? vlapl + 2*tid : dummy_vlapl;
    auto* vtau_i   = vtau   + 2*tid;
    const double lapl_a_use = traits::needs_laplacian ? lapl_i[0] : 0.0;
    const double lapl_b_use = traits::needs_laplacian ? lapl_i[1] : 0.0;
                                                         
    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], lapl_a_use, lapl_b_use, tau_i[0],
      tau_i[1], eps[tid], vrho_i[0], vrho_i[1], vsigma_i[0], vsigma_i[1], 
      vsigma_i[2], vlapl_i[0], vlapl_i[1], vtau_i[0], vtau_i[1] );

  }

}


template <typename KernelType>
__global__ MGGA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e;
  if( tid < N ) {

    const double lapl_use  = traits::needs_laplacian ? lapl[tid] : 0.0;
    traits::eval_exc_unpolar( rho[tid], sigma[tid], lapl_use, tau[tid], e );
    eps[tid] += scal_fact * e;
     

  }

}

template <typename KernelType>
__global__ MGGA_EXC_INC_GENERATOR( device_eval_exc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;
    auto* lapl_i  = traits::needs_laplacian ? (lapl + 2*tid) : lapl;
    auto* tau_i   = tau   + 2*tid;

    const double lapl_a_use = traits::needs_laplacian ? lapl_i[0] : 0.0;
    const double lapl_b_use = traits::needs_laplacian ? lapl_i[1] : 0.0;

    double e;
    traits::eval_exc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], lapl_a_use, lapl_b_use, tau_i[0], 
      tau_i[1], e );
    eps[tid] += scal_fact * e;
     

  }

}

template <typename KernelType>
__global__ MGGA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_unpolar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double e, vr, vs, vl, vt;
  if( tid < N ) {

    const double lapl_use  = traits::needs_laplacian ? lapl[tid] : 0.0;

    traits::eval_exc_vxc_unpolar( rho[tid], sigma[tid], lapl_use, tau[tid],
      e, vr, vs, vl, vt );
    eps[tid]    += scal_fact * e;
    vrho[tid]   += scal_fact * vr;
    vsigma[tid] += scal_fact * vs;
    vtau[tid]   += scal_fact * vt;
    if(traits::needs_laplacian) vlapl[tid] += scal_fact * vl;

  }

}

template <typename KernelType>
__global__ MGGA_EXC_VXC_INC_GENERATOR( device_eval_exc_vxc_inc_helper_polar_kernel ) {

  using traits = kernel_traits<KernelType>;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 

  double dummy_vlapl[2];
  if( tid < N ) {

    auto* rho_i   = rho   + 2*tid;
    auto* sigma_i = sigma + 3*tid;
    auto* lapl_i  = traits::needs_laplacian ? (lapl + 2*tid) : lapl;
    auto* tau_i   = tau   + 2*tid;

    auto* vrho_i   = vrho   + 2*tid;
    auto* vsigma_i = vsigma + 3*tid;
    auto* vlapl_i  = traits::needs_laplacian ? vlapl + 2*tid : dummy_vlapl;
    auto* vtau_i   = vtau   + 2*tid;

    const double lapl_a_use = traits::needs_laplacian ? lapl_i[0] : 0.0;
    const double lapl_b_use = traits::needs_laplacian ? lapl_i[1] : 0.0;
                                                         
                                                         
    double e, vra, vrb, vsaa,vsab,vsbb, vla, vlb, vta, vtb;
    traits::eval_exc_vxc_polar( rho_i[0], rho_i[1], sigma_i[0], 
      sigma_i[1], sigma_i[2], lapl_a_use, lapl_b_use, tau_i[0],
      tau_i[1], e, vra, vrb, vsaa, vsab, vsbb, vla, vlb, vta, vtb );

    eps[tid]    += scal_fact * e;
    vrho_i[0]   += scal_fact * vra;
    vrho_i[1]   += scal_fact * vrb;
    vsigma_i[0] += scal_fact * vsaa;
    vsigma_i[1] += scal_fact * vsab;
    vsigma_i[2] += scal_fact * vsbb;
    vtau_i[0]   += scal_fact * vta;
    vtau_i[1]   += scal_fact * vtb;
    if(traits::needs_laplacian) {
      vlapl_i[0]   += scal_fact * vla;
      vlapl_i[1]   += scal_fact * vlb;
    }

  }

}


template <typename KernelType>
LDA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, eps
  );

}

template <typename KernelType>
LDA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, eps
  );

}

template <typename KernelType>
LDA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_vxc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, eps, vxc
  );

}

template <typename KernelType>
LDA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_vxc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, eps, vxc
  );

}

template <typename KernelType>
LDA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, eps
  );

}

template <typename KernelType>
LDA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, eps
  );

}

template <typename KernelType>
LDA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_vxc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, eps, vxc
  );

}

template <typename KernelType>
LDA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_vxc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, eps, vxc
  );

}




template <typename KernelType>
GGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, eps
  );

}

template <typename KernelType>
GGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, eps
  );

}

template <typename KernelType>
GGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, eps, vrho, vsigma
  );

}

template <typename KernelType>
GGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, eps, vrho, vsigma
  );

}


template <typename KernelType>
GGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, eps
  );

}

template <typename KernelType>
GGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, eps
  );

}

template <typename KernelType>
GGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, eps, vrho, vsigma
  );

}

template <typename KernelType>
GGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, eps, vrho, vsigma
  );

}






template <typename KernelType>
MGGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, lapl, tau, eps
  );

}

template <typename KernelType>
MGGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, lapl, tau, eps
  );

}

template <typename KernelType>
MGGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, lapl, tau, eps, vrho, vsigma, vlapl, vtau
  );

}

template <typename KernelType>
MGGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    N, rho, sigma, lapl, tau, eps, vrho, vsigma, vlapl, vtau
  );

}


template <typename KernelType>
MGGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, lapl, tau, eps
  );

}

template <typename KernelType>
MGGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );
  device_eval_exc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, lapl, tau, eps
  );

}

template <typename KernelType>
MGGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_inc_helper_unpolar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, lapl, tau, eps, vrho, vsigma, vlapl, vtau
  );

}

template <typename KernelType>
MGGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar ) {

  dim3 threads(32);
  dim3 blocks( util::div_ceil( N, threads.x) );

  device_eval_exc_vxc_inc_helper_polar_kernel<KernelType><<<blocks,threads,0,stream>>>(
    scal_fact, N, rho, sigma, lapl, tau, eps, vrho, vsigma, vlapl, vtau
  );

}

#define LDA_GENERATE_DEVICE_HELPERS(KERN) \
  template LDA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar<KERN> ); \
  template LDA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar<KERN> ); \
  template LDA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar<KERN> ); \
  template LDA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar<KERN> );\
  template LDA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar<KERN> ); \
  template LDA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar<KERN> ); \
  template LDA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar<KERN> ); \
  template LDA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar<KERN> ); 

#define GGA_GENERATE_DEVICE_HELPERS(KERN) \
  template GGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar<KERN> ); \
  template GGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar<KERN> ); \
  template GGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar<KERN> ); \
  template GGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar<KERN> );\
  template GGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar<KERN> ); \
  template GGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar<KERN> ); \
  template GGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar<KERN> ); \
  template GGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar<KERN> ); 

#define MGGA_GENERATE_DEVICE_HELPERS(KERN) \
  template MGGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_unpolar<KERN> ); \
  template MGGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_unpolar<KERN> ); \
  template MGGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_unpolar<KERN> ); \
  template MGGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_unpolar<KERN> );\
  template MGGA_EXC_GENERATOR_DEVICE( device_eval_exc_helper_polar<KERN> ); \
  template MGGA_EXC_VXC_GENERATOR_DEVICE( device_eval_exc_vxc_helper_polar<KERN> ); \
  template MGGA_EXC_INC_GENERATOR_DEVICE( device_eval_exc_inc_helper_polar<KERN> ); \
  template MGGA_EXC_VXC_INC_GENERATOR_DEVICE( device_eval_exc_vxc_inc_helper_polar<KERN> ); 

LDA_GENERATE_DEVICE_HELPERS( BuiltinSlaterExchange );
LDA_GENERATE_DEVICE_HELPERS( BuiltinVWN3 );
LDA_GENERATE_DEVICE_HELPERS( BuiltinVWN_RPA );
LDA_GENERATE_DEVICE_HELPERS( BuiltinPW91_LDA );
LDA_GENERATE_DEVICE_HELPERS( BuiltinPW91_LDA_MOD );
LDA_GENERATE_DEVICE_HELPERS( BuiltinPW91_LDA_RPA );
LDA_GENERATE_DEVICE_HELPERS( BuiltinPZ81 );
LDA_GENERATE_DEVICE_HELPERS( BuiltinPZ81_MOD );

GGA_GENERATE_DEVICE_HELPERS( BuiltinB88   );
GGA_GENERATE_DEVICE_HELPERS( BuiltinLYP   );
GGA_GENERATE_DEVICE_HELPERS( BuiltinPBE_X );
GGA_GENERATE_DEVICE_HELPERS( BuiltinRevPBE_X );
GGA_GENERATE_DEVICE_HELPERS( BuiltinPBE_C );

GGA_GENERATE_DEVICE_HELPERS( BuiltinB3LYP );
GGA_GENERATE_DEVICE_HELPERS( BuiltinPBE0  );

MGGA_GENERATE_DEVICE_HELPERS( BuiltinSCAN_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinSCAN_C );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinR2SCAN_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinR2SCAN_C );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinFT98_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinM062X_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinM062X_C );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinPKZB_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinPKZB_C );

MGGA_GENERATE_DEVICE_HELPERS( BuiltinPC07_K );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinPC07OPT_K );

MGGA_GENERATE_DEVICE_HELPERS( BuiltinSCANL_C );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinSCANL_X );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinR2SCANL_C );
MGGA_GENERATE_DEVICE_HELPERS( BuiltinR2SCANL_X );

LDA_GENERATE_DEVICE_HELPERS( BuiltinEPC17_1 )
LDA_GENERATE_DEVICE_HELPERS( BuiltinEPC17_2 )
LDA_GENERATE_DEVICE_HELPERS( BuiltinEPC18_1 )
LDA_GENERATE_DEVICE_HELPERS( BuiltinEPC18_2 )

}
}

